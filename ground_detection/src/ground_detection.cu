#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include "ground_detection/ground_detection.hpp"

#define BLOCK_DIM 128

__device__ float3 operator-(float3 a, float3 b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 operator/(float3 a, float b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__device__ float3 cross(const float3& a, const float3& b) {
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

__device__ float dot(const float3& a, const float3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float distance(const float3& p, const float3& n, float d) {
    return fabs(dot(p, n) + d) / sqrtf(n.x * n.x + n.y * n.y + n.z * n.z);
}

__device__ void atomicUpdate(int* address, int val_to_compare_and_store, float4* float4_address, float4 val_to_store)
{

    int ret = *address;

    while(val_to_compare_and_store > ret) {

        int old = ret;
        ret = atomicCAS(address, old, val_to_compare_and_store);
        if(ret == old) { // it means that the current thread has modified the value and returned the old value
            *float4_address = val_to_store;
            break;
        }
    }
}

__global__ void ransac(const float4*  pointCloud, int numPoints, float4* bestPlane, int* inliers, float distanceThreshold, int maxIterations) {
    
    __shared__ float s_bestPlane[BLOCK_DIM]; // Shared memory for storing each thread's best plane parameters
    s_bestPlane[threadIdx.x] = 0; // Initialize the number of inliers for the current thread
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx >= maxIterations) return;

    hiprandState state;
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);

    int index[3];
    for (int i = 0; i < 3; ++i)
        index[i] = hiprand(&state) % numPoints;

    float3 p1 = make_float3( pointCloud[index[0]].x,  pointCloud[index[0]].y,  pointCloud[index[0]].z);
    float3 p2 = make_float3( pointCloud[index[1]].x,  pointCloud[index[1]].y,  pointCloud[index[1]].z);
    float3 p3 = make_float3( pointCloud[index[2]].x,  pointCloud[index[2]].y,  pointCloud[index[2]].z);

    float3 normal = cross(p2 - p1, p3 - p1);
    float length = sqrtf(dot(normal, normal));
    normal = normal / length;

    float d = -dot(normal, p1);

    int numInliers = 0;

    for (int i = 0; i < numPoints; ++i) {
        auto& point = pointCloud[i];
        float dist = distance(make_float3( point.x,  point.y,  point.z), normal, d);
        if (dist <= distanceThreshold)
            numInliers++;
    }

    s_bestPlane[threadIdx.x] = numInliers; // Store the number of inliers for this thread in shared memory

    __syncthreads(); // Ensure all threads have stored their number of inliers before reduction

    // Reduction to find the best solution among all threads of the same block
    // This solution allows to reduce the number of threads trying to update values on memory.
    if (threadIdx.x == 0) {
        double maxInliers = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            if (s_bestPlane[i] > maxInliers) {
                maxInliers = s_bestPlane[i];
            }
        }
        float4 current_plane = make_float4(normal.x, normal.y, normal.z, d);
        atomicUpdate(&inliers[0], numInliers, bestPlane, current_plane);
    }
}

__global__ void getInliersMask(const float4* pointCloud, int numPoints, const float4* bestPlane, int* inliersMask, float distanceThreshold) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= numPoints) return;

    inliersMask[idx] = 0; // pre-set all the values of the inliersMask to 0.

    auto& point = pointCloud[idx];
    float3 normal = make_float3(bestPlane->x, bestPlane->y, bestPlane->z);
    float d = bestPlane->w;
    float dist = distance(make_float3( point.x, point.y, point.z), normal, d);
    if (dist <= distanceThreshold)
        inliersMask[idx] = 1;

}

void groundPointsDetection(const float4* d_point_cloud, size_t num_current_points, int* d_inliers_mask, float4* d_fitting_plane, int* num_inliers, float distance_threshold, int max_iterations) {
    int inputSize = num_current_points; // The size of the input data
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    int threadsPerBlock = BLOCK_DIM;

    int blocksPerGrid = inputSize / threadsPerBlock + ((inputSize % threadsPerBlock) ? 1:0); 
    
    printf("# blocks: %d # threads: %d", blocksPerGrid, threadsPerBlock);
    ransac<<<blocksPerGrid, threadsPerBlock>>>(d_point_cloud, num_current_points, d_fitting_plane, num_inliers, distance_threshold, max_iterations);
    gpuErrchk( hipPeekAtLastError() );

    getInliersMask<<<blocksPerGrid, threadsPerBlock>>>(d_point_cloud, num_current_points, d_fitting_plane, d_inliers_mask, distance_threshold);
    gpuErrchk( hipPeekAtLastError() );
}

/*
// For test purpose
// TODO: Use in unit tests
std::vector<float4> generatePointCloudOnPlane(int numPoints, double planeHeight) {
    std::vector<float4> points;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(-10.0, 10.0); // Adjust range as needed
    std::uniform_real_distribution<double> dis_z(-0.05, 0.05); // Adjust range as needed

    for (int i = 0; i < numPoints; ++i) {
        float4 p;
        p.x = dis(gen);
        p.y = dis(gen);
        p.z = planeHeight + dis_z(gen); // All points are on the same plane parallel to XY plane
        points.push_back(p);
    }

    return points;
}

int main() {
    std::vector<float4> points = generatePointCloudOnPlane(100000, 0);//{{1, 2, 0.1}, {4, 5, 0.05}, {7, 8, 0.01}, {10, 11, 10.002}, {13, 14, 10.015}};
    int numPoints = points.size();

    float distanceThreshold = 0.1f;
    int maxIterations = 10000;
    
    float4* d_points;
    float4* d_bestPlane;
    int* d_inliers;

    hipMalloc((void**)&d_points, numPoints * sizeof(float4));
    hipMemcpy(d_points, points.data(), numPoints * sizeof(float4), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_bestPlane, sizeof(float4)); // One for normal, one for d
    hipMalloc((void**)&d_inliers, sizeof(int));

    int* inliers = new int;
    *inliers = 0;

    hipMemcpy(d_inliers, inliers, sizeof(int), hipMemcpyHostToDevice);

    int* d_inliers_mask;
    hipMalloc((void**)&d_inliers_mask, numPoints * sizeof(int));

    int threadsPerBlock = BLOCK_DIM;
    int blocksPerGrid = (maxIterations + threadsPerBlock - 1) / threadsPerBlock;

    // Use APIs for thread occupancy
    //===============================
    // int numBlocks;        // Occupancy in terms of active blocks
    // int blockSize = 1024;

    // // These variables are used to convert occupancy to warps
    // int device;
    // hipDeviceProp_t prop;
    // int activeWarps;
    // int maxWarps;

    // hipGetDevice(&device);
    // hipGetDeviceProperties(&prop, device);

    // hipOccupancyMaxActiveBlocksPerMultiprocessor(
    //     &numBlocks,
    //     ransac,
    //     blockSize,
    //     0);

    // activeWarps = numBlocks * blockSize / prop.warpSize;
    // maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
    // std::cout << "Max active blocks: " << numBlocks << std::endl;
    // std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;
    // std::cout << "Max warps:: " << maxWarps << std::endl;
    //===============================
    ///

    ransac<<<blocksPerGrid, threadsPerBlock>>>(d_points, numPoints, d_bestPlane, d_inliers, distanceThreshold, maxIterations);
    gpuErrchk( hipPeekAtLastError() );

    threadsPerBlock = BLOCK_DIM;
    blocksPerGrid = (numPoints + threadsPerBlock - 1) / threadsPerBlock;

    getInliersMask<<<blocksPerGrid, threadsPerBlock>>>(d_points, numPoints, d_bestPlane, d_inliers_mask, distanceThreshold);
    gpuErrchk( hipPeekAtLastError() );

    hipMemcpy(inliers, d_inliers, sizeof(int), hipMemcpyDeviceToHost);

    float4 bestPlane[1];
    hipMemcpy(bestPlane, d_bestPlane, sizeof(float4), hipMemcpyDeviceToHost);

    int inliersMask[numPoints];
    hipMemcpy(inliersMask, d_inliers_mask, numPoints * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Best plane parameters: Normal = (" << bestPlane[0].x << ", " << bestPlane[0].y << ", " << bestPlane[0].z << "), d = " << bestPlane[0].w << std::endl;

    std::cout << "inliers: " << inliers[0] << std::endl;

    bool isWorking = true;
    for(int i=0; i<numPoints; i++) {
        if(inliersMask[i] != 1) {
            isWorking=false;
            std::cout << "Not working at idx: " << i << std::endl;
        };
    }

    std::cout << "is working: " << isWorking << std::endl;
    
    hipFree(d_points);
    hipFree(d_bestPlane);
    hipFree(d_inliers);
    delete inliers;

    return 0;
}
*/
